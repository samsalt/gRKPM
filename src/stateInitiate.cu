#include "rkpmProlblem.h"

void grkpm::stateInitiate()
{
    int i=0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);


    blocksPerGrid = (nc + threadsPerBlock - 1) / threadsPerBlock;

    EchoVar("blocksPerGrid",blocksPerGrid);

    for (int i =0; i< simulationParameter.blockNum; i++)
    {
           simulationParameter.blockInfo[i].formCmat();
    }

    err = hipMalloc(&dspDev, nc * sizeof(cellDsp));
    except(err,"Fail to allocate device memory, dspDev");
    err = hipMalloc(&positionDev, nc * sizeof(cellPosition));
    except(err,"Fail to allocate device memory, position");
    err = hipMalloc(&forceDev, nc * sizeof(cellForce));
    except(err,"Fail to allocate device memory, force");
    err = hipMalloc(&nodeNeighbor, nc * sizeof(gmNodeNeighbor));
    except(err,"Fail to allocate device memory, nodeNeighbor");

    err = hipMemcpy(dspDev, hostDsp, nc * sizeof(cellDsp), hipMemcpyHostToDevice);
    except(err,"Fail to tansfer data to device, dsp");
    err = hipMemcpy(positionDev, hostPosition, nc * sizeof(cellPosition), hipMemcpyHostToDevice);
    except(err,"Fail to tansfer data to device, position");
    err = hipMemcpy(forceDev, hostForce, nc * sizeof(cellForce), hipMemcpyHostToDevice);
    except(err,"Fail to tansfer data to device, force");

    err = hipMalloc(&(essentialNodeDev), simulationParameter.sideSetNum * sizeof(int*));
    except(err,"Fail to allocate device memory, essential");

    for (int i=0; i<simulationParameter.sideSetNum;i++)
    {
        err = hipMalloc(&(essentialNodeDev[i]), nodeNumInSet[i] * sizeof(int));
        except(err,"Fail to allocate device memory, essential node set");
        err = hipMemcpy(essentialNodeDev[i], hostEssentialNode[i], nodeNumInSet[i]  * sizeof(int), hipMemcpyHostToDevice);
        except(err,"Fail to tansfer data to device, essential");
        free(hostEssentialNode[i]);
    }
    free(hostEssentialNode);

}
